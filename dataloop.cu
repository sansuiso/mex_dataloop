#include "hip/hip_runtime.h"
#include "dataloop.h"

__global__ void dataloop(float *src, float *dest) 
{
	int tid = blockIdx.x;
	dest[tid] = src[tid];
}

void process_data_with_cuda(float *host_src, float *host_dest, int N)
{
	float *d_src = NULL;
	float *d_dest = NULL;

	memset(host_dest, 0, N*sizeof(float));

	// Allocate on device
	hipMalloc((void**)&d_src, N*sizeof(float));
	hipMalloc((void**)&d_dest, N*sizeof(float));

	// Transfer src to device
	hipMemcpy(d_src, host_src, N*sizeof(float), hipMemcpyHostToDevice);

	// Launch kernel
	dataloop<<<N, 1>>>(d_src, d_dest);

	// Fetch data back
	hipMemcpy(host_dest, d_dest, N*sizeof(float), hipMemcpyDeviceToHost);

	// Release memory
	hipFree(d_src);
	hipFree(d_dest);
}